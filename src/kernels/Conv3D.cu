#include "hip/hip_runtime.h"

#include <core/device/CudaObject.hpp>
#include <kernels/Conv3D.hpp>

#include <cstddef>
#include <cstdint>
#include <iostream>

#include <core/device/DeviceTensor.hpp>

namespace kernels
{
    __global__ void image_convolution(float* input, float* output, float* kernel, 
                                    ConvolutionScalarData scalar)
    {
        int w = blockIdx.x * blockDim.x + threadIdx.x;  // width
        int h = blockIdx.y * blockDim.y + threadIdx.y;  // height
        int c = blockIdx.z; // channel

        const std::size_t input_h = scalar.input_h;
        const std::size_t input_w = scalar.input_w;
        const std::size_t kernel_dim = scalar.kernel_dim;
        const std::size_t channels = scalar.channels;
        const std::size_t padding = 0;  // Not supported atm
        const std::size_t dilation = scalar.dilation;
        const std::size_t stride = scalar.stride;

        const std::size_t output_dim = ((input_h + 2*padding - dilation * (kernel_dim-1)-1)/stride)+1;
    
        //Check if thread is out of bounds
        if (w >= scalar.get_output_width() || h >= scalar.get_output_height() || c >= channels) return;

        float kernel_sum{};
        for (std::size_t kh = 0; kh < kernel_dim; ++kh)
        {
            for (std::size_t kw = 0; kw < kernel_dim; ++kw)
            {
                kernel_sum += kernel[(kh * kernel_dim + kw) * channels + c];
            }
        }

        float max_possible_val = kernel_sum * 255.0f;
        if (fabs(max_possible_val) < 1e-5f) max_possible_val = 1.0f;  // avoid div/0
        const std::size_t min_possible_val = 0;
        float sum{};

        // Run through kernel
        for (std::size_t kernel_h = 0; kernel_h < kernel_dim; ++kernel_h) // For each kernel row
        {
            for (std::size_t kernel_w = 0; kernel_w < kernel_dim; ++kernel_w) // For each kernel value
            {
                int current_h = h * stride + kernel_h * dilation - padding; // Compute the corresponding input row 
                int current_w = w * stride + kernel_w * dilation - padding; // Compute the corresponding input column 
                
                // Bounds check to avoid out-of-bounds memory access.
                if (current_h >= 0 && current_h < input_h && current_w >= 0 && current_w < (int)input_w)
                {
                    // Compute the linear index into the input array (HWC format)
                    std::size_t input_idx = (current_h * input_w + current_w) * channels + c;
                    // Compute the linear index into the kernel array (HWC format)
                    std::size_t kernel_idx = (kernel_h * kernel_dim + kernel_w) * channels + c;
                    // Accumulate the convolution result
                    sum += input[input_idx] * kernel[kernel_idx];
                }
            }
        }
        if (w < output_dim && h < output_dim && c < channels)
        {
            //float normalized_val = static_cast<float>(255.0f * (sum - min_possible_val) / (max_possible_val - min_possible_val));
            std::size_t out_idx = (h * scalar.get_output_width() + w) * channels + c;
            float gain = 25.0f;
            output[out_idx] = min(max(sum * gain, 0.0f), 255.0f);
        }
    }

    __host__ core::Tensor launch_conv3d_kernel(
            const core::Tensor& input, 
            const core::Tensor& kernel, 
            ConvolutionScalarData scalar
        )
    {
        core::device::DeviceTensor cu_input{input};
        core::device::DeviceTensor cu_kernel{kernel};
        
        int out_height = scalar.get_output_height();
        int out_width  = scalar.get_output_width();
        
        core::Tensor out_tensor{out_height, out_width, scalar.channels};
        core::device::DeviceTensor cu_output{out_tensor};

        dim3 blockDim(16, 16);
        dim3 gridDim((out_width + blockDim.x - 1) / blockDim.x,
                    (out_height + blockDim.y - 1) / blockDim.y,
                    scalar.channels);

        if (scalar.channels <= 0 || out_width <= 0 || out_height <= 0) {
            std::cerr << "Invalid kernel launch dimensions: "
                    << "channels=" << scalar.channels
                    << ", out_width=" << out_width
                    << ", out_height=" << out_height << "\n";
            std::exit(EXIT_FAILURE);
        }

        image_convolution<<<gridDim, blockDim>>>(cu_input.get_device(), cu_output.get_device(), cu_kernel.get_device(), scalar);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) 
        {
            std::cerr << "Kernel launch error: " << hipGetErrorString(err) << "\n";
            std::exit(EXIT_FAILURE);
        }
        
        hipError_t sync_err = hipDeviceSynchronize();
        if (sync_err != hipSuccess) {
            std::cerr << "CUDA sync error: " << hipGetErrorString(sync_err) << "\n";
            std::exit(EXIT_FAILURE);
        }

        cu_output.sync_to_host();

        auto ret = *dynamic_cast<core::Tensor*>(&cu_output);
        return ret;
    }
}

